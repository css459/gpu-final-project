#include <stdio.h>
#include "cuda_util.h"
#include "sampling.h"

int main() {

    printf("GPU Mem: %zu\n", get_global_memory_size_for_device(0));
    printf("Samples Possible: %d\n", get_sample_size_for_device(0, 20, sizeof(float)));

    // Set the Random Seed to Time
    srand((unsigned int) time(NULL));

    SamplingProperties props = make_properties(
            "test_data/xy.csv",        // csv_file_path
            11209389,       // file_size
            3000000,         // line_buffer_size
            1000,           // random_chunk_size
            21,             // elements_per_line
            sizeof(float),  // element_size_bytes
            4               // cuda_device_count
            );

    float** cuda_samples = load_devices(&props);

    for (int i = 0; i < props.cuda_device_count; i++) hipFree((void *) cuda_samples[i]);
    free(cuda_samples);

    return 0;
}
