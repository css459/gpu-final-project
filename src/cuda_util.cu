//
// Created by cole on 11/8/18.
//

#include <hip/hip_runtime.h>
#include <stdio.h>
#include "cuda_util.h"

// Method to check for CUDA errors
#define cudaCheckError(err) {                                                                    \
    if (err != hipSuccess) {                                                                    \
        fprintf(stderr,"[ ERR ] CUDA: %s %s %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
        exit(1);                                                                                 \
    }                                                                                            \
}

int get_num_cuda_devices() {
    int device_count;
    hipError_t err = hipGetDeviceCount(&device_count);
    cudaCheckError(err);

    if (device_count < 1) {
        printf("[ ERR ] No CUDA devices, aborting.\n");
        exit(1);
    }

    return device_count;
}

size_t get_global_memory_size_for_device(int device_number) {
    hipDeviceProp_t dev;
    hipError_t err = hipGetDeviceProperties(&dev, 0);
    cudaCheckError(err);

    return dev.totalGlobalMem;

}